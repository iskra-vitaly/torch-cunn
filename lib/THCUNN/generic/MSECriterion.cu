
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MSECriterion.cu"
#else

void THNN_(MSECriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage)
{
  THCUNN_assertSameGPU_generic(state, 2, input, target);
  THArgCheck(THCTensor_(nElement)(state, input) == THCTensor_(nElement)(state, target), 2,
    "input and target need to have the same number of elements"
  );

  long size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  accreal sum = thrust::inner_product(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, (accreal) 0,
    thrust::plus<accreal>(), mse_functor<real, accreal>());

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}

void THNN_(MSECriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage)
{
  THCUNN_assertSameGPU_generic(state, 3, input, target, gradInput);
  THArgCheck(THCTensor_(nElement)(state, input) == THCTensor_(nElement)(state, target), 2,
    "input and target need to have the same number of elements"
  );

  long size = THCTensor_(nElement)(state, input);
  accreal norm = sizeAverage ? (accreal)(2)/size : (accreal)(2);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, gradInput_data,
    mse_updateGradInput_functor<real, accreal>(norm));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif